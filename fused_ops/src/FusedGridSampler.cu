#include "hip/hip_runtime.h"
// #define TORCH_ASSERT_NO_OPERATORS
#include <ATen/OpMathType.h>
#include <ATen/native/GridSamplerUtils.h>
#include <ATen/native/cuda/GridSampler.cuh>
#include <ATen/native/cuda/UpSample.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/Atomic.cuh>
#include <c10/cuda/CUDAStream.h>
#include <c10/cuda/CUDAGuard.h>
// #include <ATen/native/cuda/KernelUtils.h>
// #include <ATen/core/TensorBase.h>
#include <ATen/Dispatch.h>
#include <c10/macros/Macros.h>
#include <torch/torch.h>
#include <cmath>
#include <iostream>
#include "common.h"

// Core CUDA headers
#include <hip/hip_runtime.h>
#include <torch/extension.h>
// PyTorch CUDA headers

using namespace at::cuda::detail;
using namespace at::native;
using at::native::detail::GridSamplerInterpolation;
using at::native::detail::GridSamplerPadding;

inline void check_grid_sampler_common_v2(
  const torch::Tensor& input,
  const torch::Tensor& grid
) {
  auto input_opt = input.options();
  auto grid_opt = grid.options();

  TORCH_CHECK(
    input.defined(),
    "grid_sampler(): expected input to not be undefined");
  TORCH_CHECK(
    grid.defined(),
    "grid_sampler(): expected grid to not be undefined");
  TORCH_CHECK(
    input_opt.device() == grid_opt.device(),
    "grid_sampler(): expected input and grid to be on same device, but input "
    "is on ", input_opt.device(), " and grid is on ", grid_opt.device());
//   TORCH_CHECK(
//     input_opt.layout() == kStrided && grid_opt.layout() == kStrided,
//     "grid_sampler(): expected input and grid to have torch.strided layout, but "
//     "input has ", input_opt.layout(), " and grid has ", grid_opt.layout());
    // removed this because we can now work with broadcasted input and grid
//   TORCH_CHECK(
//     input.size(0) == grid.size(0),
//     "grid_sampler(): expected grid and input to have same batch size, but got "
//     "input with sizes ", input.sizes(), " and grid with sizes ", grid.sizes());
  TORCH_CHECK(
    grid.size(-1) == input.dim() - 2,
    "grid_sampler(): expected grid to have size ", input.dim() - 2, " in last "
    "dimension, but got grid with sizes ", grid.sizes());
//   for (const auto i : c10::irange(2, input.dim())) {
 for (int i = 2; i < input.dim(); i++) {
    TORCH_CHECK(input.size(i) > 0,
      "grid_sampler(): expected input to have non-empty spatial dimensions, "
      "but input has sizes ", input.sizes(), " with dimension ", i, " being "
      "empty");
  }
}

template<typename scalar_t, typename index_t>
__forceinline__ __device__
void safe_add_3d_oneoffset(
                scalar_t *data, 
                int d, int h, int w,
                int D, int H, int W,
                scalar_t delta,
                const index_t NC_offset,
                const index_t memory_span) {
  if (within_bounds_3d(d, h, w, D, H, W)) {
    gpuAtomicAdd(data + NC_offset + w + W * (h + H * d), delta);
  }
}

template <typename scalar_t, typename grid_t, typename index_t>
C10_LAUNCH_BOUNDS_1(BLOCKSIZE_3D)
__global__ void fused_grid_sampler_3d_forward_kernel(
    const index_t count,
    const scalar_t* input,
    const grid_t* grid,
    const grid_t* affine_3d,
    const index_t N,
    const index_t C,
    const index_t Di,
    const index_t Hi,
    const index_t Wi,
    const index_t D,
    const index_t H,
    const index_t W,
    const float grid_xmin,
    const float grid_ymin,
    const float grid_zmin,
    const float grid_xmax,
    const float grid_ymax,
    const float grid_zmax,
    const bool is_displacement,
    scalar_t* output,
    const GridSamplerInterpolation interpolation_mode,
    const GridSamplerPadding padding_mode,
    const bool align_corners,
    const bool broadcast_input,
    const bool broadcast_affine_3d,
    const bool broadcast_grid
    ) {

    // define datatypes to operate
    using opmath_t = at::opmath_type<scalar_t>;
    using gridmath_t = at::opmath_type<grid_t>;

    CUDA_KERNEL_LOOP_TYPE(index, count, index_t) {
        const index_t w = index % W;
        const index_t h = (index / W) % H;
        const index_t d = (index / (H * W)) % D;
        const index_t n = index / (D * H * W);
        // we have 3 coordinates for each grid point, so we multiply the index by 3
        const index_t grid_offset = 3 * (w + W * (h + H * (d + (broadcast_grid ? 0 : (D * n)))));

        // this is only affine coordinate
        gridmath_t ix = 0, iy = 0, iz = 0;
        gridmath_t x = 0, y = 0, z = 0;
        if (!grid) {
            // if grid is not provided, then affine matrix is multiplied to input coordinate
            // displacement is ignored
            // just affine coordiante here, we load the entire affine matrix
            const grid_t* affine_3d_ptr = affine_3d + (broadcast_affine_3d ? 0 : (12 * n));
            // get normalized coordinate
            ix = w * (grid_xmax - grid_xmin) / (W-1) + grid_xmin;
            iy = h * (grid_ymax - grid_ymin) / (H-1) + grid_ymin;
            iz = d * (grid_zmax - grid_zmin) / (D-1) + grid_zmin;
            // apply affine matrix
            x = affine_3d_ptr[0] * ix + affine_3d_ptr[1] * iy + affine_3d_ptr[2] * iz + affine_3d_ptr[3];
            y = affine_3d_ptr[4] * ix + affine_3d_ptr[5] * iy + affine_3d_ptr[6] * iz + affine_3d_ptr[7];
            z = affine_3d_ptr[8] * ix + affine_3d_ptr[9] * iy + affine_3d_ptr[10] * iz + affine_3d_ptr[11];
        }
        else {
            // grid is provided, load the grid coordinate
            // x = grid[grid_offset];
            // y = grid[grid_offset + 1];
            // z = grid[grid_offset + 2];
            // if these are warp coordinates (`is_displacement` is false), affine matrix is ignored
            // if these are displacement coordinates, get the grid coordinates x, y, z, multiply by affine matrix, then add to displacement
            if (is_displacement) {
                // get grid coordinate
                ix = w * (grid_xmax - grid_xmin) / (W-1) + grid_xmin;
                iy = h * (grid_ymax - grid_ymin) / (H-1) + grid_ymin;
                iz = d * (grid_zmax - grid_zmin) / (D-1) + grid_zmin;
                // apply affine matrix
                if(affine_3d) {
                    const grid_t* affine_3d_ptr = affine_3d + (broadcast_affine_3d ? 0 : (12 * n));
                    x = affine_3d_ptr[0] * ix + affine_3d_ptr[1] * iy + affine_3d_ptr[2] * iz + affine_3d_ptr[3];
                    y = affine_3d_ptr[4] * ix + affine_3d_ptr[5] * iy + affine_3d_ptr[6] * iz + affine_3d_ptr[7];
                    z = affine_3d_ptr[8] * ix + affine_3d_ptr[9] * iy + affine_3d_ptr[10] * iz + affine_3d_ptr[11];
                }
                else {
                    x = ix; y = iy; z = iz;
                }
                // add to displacement
                x += grid[grid_offset];
                y += grid[grid_offset + 1];
                z += grid[grid_offset + 2];
            }
            else {
                // just get warp
                x = grid[grid_offset];
                y = grid[grid_offset + 1];
                z = grid[grid_offset + 2];
            }
        }

        // get the corresponding input x, y, z co-ordinates from grid
        ix = grid_sampler_compute_source_index(x, Wi, padding_mode, align_corners);
        iy = grid_sampler_compute_source_index(y, Hi, padding_mode, align_corners);
        iz = grid_sampler_compute_source_index(z, Di, padding_mode, align_corners);

        if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
            // get corner pixel values from (x, y, z)
            // for 4d, we used north-east-south-west
            // for 5d, we add top-bottom
            index_t ix_tnw = static_cast<index_t>(::floor(ix));
            index_t iy_tnw = static_cast<index_t>(::floor(iy));
            index_t iz_tnw = static_cast<index_t>(::floor(iz));

            index_t ix_tne = ix_tnw + 1;
            index_t iy_tne = iy_tnw;
            index_t iz_tne = iz_tnw;

            index_t ix_tsw = ix_tnw;
            index_t iy_tsw = iy_tnw + 1;
            index_t iz_tsw = iz_tnw;

            index_t ix_tse = ix_tnw + 1;
            index_t iy_tse = iy_tnw + 1;
            index_t iz_tse = iz_tnw;

            index_t ix_bnw = ix_tnw;
            index_t iy_bnw = iy_tnw;
            index_t iz_bnw = iz_tnw + 1;

            index_t ix_bne = ix_tnw + 1;
            index_t iy_bne = iy_tnw;
            index_t iz_bne = iz_tnw + 1;

            index_t ix_bsw = ix_tnw;
            index_t iy_bsw = iy_tnw + 1;
            index_t iz_bsw = iz_tnw + 1;

            index_t ix_bse = ix_tnw + 1;
            index_t iy_bse = iy_tnw + 1;
            index_t iz_bse = iz_tnw + 1;

            // get surfaces to each neighbor:
            scalar_t tnw = (ix_bse - ix)    * (iy_bse - iy)    * (iz_bse - iz);
            scalar_t tne = (ix    - ix_bsw) * (iy_bsw - iy)    * (iz_bsw - iz);
            scalar_t tsw = (ix_bne - ix)    * (iy    - iy_bne) * (iz_bne - iz);
            scalar_t tse = (ix    - ix_bnw) * (iy    - iy_bnw) * (iz_bnw - iz);
            scalar_t bnw = (ix_tse - ix)    * (iy_tse - iy)    * (iz - iz_tse);
            scalar_t bne = (ix    - ix_tsw) * (iy_tsw - iy)    * (iz - iz_tsw);
            scalar_t bsw = (ix_tne - ix)    * (iy    - iy_tne) * (iz - iz_tne);
            scalar_t bse = (ix    - ix_tnw) * (iy    - iy_tnw) * (iz - iz_tnw);

            // get input and output strides
            index_t out_sC = D * H * W;
            index_t inp_sC = Di * Hi * Wi;
            // get input and output pointers
            const scalar_t* inp_ptr_NC = input + (broadcast_input ? 0 : (n * (C * inp_sC)));
            scalar_t* out_ptr_NCDHW = output + (w + W * (h + H * (d + D * C * n))); // add batch, depth, height, width offset

            for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCDHW += out_sC) {
                //   (c, iz_tnw, iy_tnw, ix_tnw) * tnw + (c, iz_tne, iy_tne, ix_tne) * tne
                // + (c, iz_tsw, iy_tsw, ix_tsw) * tsw + (c, iz_tse, iy_tse, ix_tse) * tse
                // + (c, iz_bnw, iy_bnw, ix_bnw) * bnw + (c, iz_bne, iy_bne, ix_bne) * bne
                // + (c, iz_bsw, iy_bsw, ix_bsw) * bsw + (c, iz_bse, iy_bse, ix_bse) * bse
                scalar_t out_acc = 0;
                if (within_bounds_3d(iz_tnw, iy_tnw, ix_tnw, Di, Hi, Wi)) {
                    out_acc += inp_ptr_NC[ix_tnw + Wi * (iy_tnw + Hi * iz_tnw)] * tnw;
                }
                if (within_bounds_3d(iz_tne, iy_tne, ix_tne, Di, Hi, Wi)) {
                    out_acc += inp_ptr_NC[ix_tne + Wi * (iy_tne + Hi * iz_tne)] * tne;
                }
                if (within_bounds_3d(iz_tsw, iy_tsw, ix_tsw, Di, Hi, Wi)) {
                    out_acc += inp_ptr_NC[ix_tsw + Wi * (iy_tsw + Hi * iz_tsw)] * tsw;
                }
                if (within_bounds_3d(iz_tse, iy_tse, ix_tse, Di, Hi, Wi)) {
                    out_acc += inp_ptr_NC[ix_tse + Wi * (iy_tse + Hi * iz_tse)] * tse;
                }
                if (within_bounds_3d(iz_bnw, iy_bnw, ix_bnw, Di, Hi, Wi)) {
                    out_acc += inp_ptr_NC[ix_bnw + Wi * (iy_bnw + Hi * iz_bnw)] * bnw;
                }
                if (within_bounds_3d(iz_bne, iy_bne, ix_bne, Di, Hi, Wi)) {
                    out_acc += inp_ptr_NC[ix_bne + Wi * (iy_bne + Hi * iz_bne)] * bne;
                }
                if (within_bounds_3d(iz_bsw, iy_bsw, ix_bsw, Di, Hi, Wi)) {
                    out_acc += inp_ptr_NC[ix_bsw + Wi * (iy_bsw + Hi * iz_bsw)] * bsw;
                }
                if (within_bounds_3d(iz_bse, iy_bse, ix_bse, Di, Hi, Wi)) {
                    out_acc += inp_ptr_NC[ix_bse + Wi * (iy_bse + Hi * iz_bse)] * bse;
                }
                *out_ptr_NCDHW = out_acc;
            }
        } else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
            index_t ix_nearest = static_cast<index_t>(std::nearbyint(ix));
            index_t iy_nearest = static_cast<index_t>(std::nearbyint(iy));
            index_t iz_nearest = static_cast<index_t>(std::nearbyint(iz));

            index_t out_sC = D * H * W;
            index_t inp_sC = Di * Hi * Wi;

            // assign nearest neighbour pixel value to output pixel
            const scalar_t* inp_ptr_NC = input + (broadcast_input ? 0 : (n * (C * inp_sC)));
            scalar_t* out_ptr_NCDHW = output + (w + W * (h + H * (d + D * C * n))); // add batch, depth, height, width offset

            for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCDHW += out_sC) {
                if (within_bounds_3d(iz_nearest, iy_nearest, ix_nearest, Di, Hi, Wi)) {
                    *out_ptr_NCDHW = inp_ptr_NC[ix_nearest + Wi * (iy_nearest + Hi * iz_nearest)];
                } else {
                    *out_ptr_NCDHW = static_cast<scalar_t>(0);
                }
            }
        }
    }
}

// Note [Passing pointer and offset to fastAtomicAdd]
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
// For its internal bounds checking, fastAtomicAdd needs to know where the destination address
// lies relative to the entire tensor, so we pass the base grad_input.data and full offset information,
// including batch * channel offset (NC_offset).
template <typename scalar_t, typename grid_t, typename index_t>
C10_LAUNCH_BOUNDS_1(BLOCKSIZE_3D)
__global__ void fused_grid_sampler_3d_backward_kernel(
        const index_t count, /* D * H * W */
        const scalar_t* input,
        const grid_t* grid,
        const grid_t* affine_3d,
        // grads
        const scalar_t* grad_output,
        scalar_t* grad_input,
        grid_t* grad_affine_collect,
        grid_t* grad_grid,
        // input size parameters
        const index_t N,
        const index_t C,
        const index_t Di,
        const index_t Hi,
        const index_t Wi,
        const index_t D,
        const index_t H,
        const index_t W,
        const float grid_xmin,
        const float grid_ymin,
        const float grid_zmin,
        const float grid_xmax,
        const float grid_ymax,
        const float grid_zmax,
        const bool is_displacement,
        const GridSamplerInterpolation interpolation_mode,
        const GridSamplerPadding padding_mode,
        const bool align_corners,
        const bool broadcast_input,
        const bool broadcast_affine_3d,
        const bool broadcast_grid) {

    // batch index is separated from the other dimensions
    const index_t n = blockIdx.y;

    using opmath_t = at::opmath_type<scalar_t>;
    using gridmath_t = at::opmath_type<grid_t>;

    // collect affine gradients
    gridmath_t _affine_grad_[12];
    #pragma unroll
    for(index_t i = 0; i < 12; ++i) {
        _affine_grad_[i] = 0;
    }
    // shared memory to take affine gradient sum over block
    __shared__ gridmath_t _affine_grad_shared_[BLOCKSIZE_3D];   
    _affine_grad_shared_[threadIdx.x] = 0;

    // also collect affine map locally to avoid loading multiple times
    gridmath_t _affine_map_[12];
    if (affine_3d) {
        const index_t offset = broadcast_affine_3d ? 0 : (12 * n);
        #pragma unroll
        for (index_t i = 0; i < 12; ++i) {
            _affine_map_[i] = affine_3d[offset + i];
        }
    }

    // loop over
    CUDA_KERNEL_LOOP_TYPE(index, count, index_t) {
        const index_t w = index % W;
        const index_t h = (index / W) % H;
        const index_t d = (index / (H * W)) % D;
        const index_t grid_offset = 3 * (w + W * (h + H * (d + (broadcast_grid ? 0 : (D * n)))));

        // get the corresponding input x, y, z co-ordinates from grid
        gridmath_t pax, pay, paz;   // pax = pre-affine x
        gridmath_t ix, iy, iz;
        gridmath_t x, y, z;

        // get grid coordinates  phi = (A? * x + u?)
        if (!grid) {
            // if grid is not provided, then affine matrix is multiplied to input coordinate
            // displacement is ignored
            // just affine coordiante here, we load the entire affine matrix
            // get normalized coordinate
            ix = w * (grid_xmax - grid_xmin) / (W-1) + grid_xmin;
            iy = h * (grid_ymax - grid_ymin) / (H-1) + grid_ymin;
            iz = d * (grid_zmax - grid_zmin) / (D-1) + grid_zmin;
            pax = ix; pay = iy; paz = iz;
            // apply affine matrix
            x = _affine_map_[0] * ix + _affine_map_[1] * iy + _affine_map_[2] * iz + _affine_map_[3];
            y = _affine_map_[4] * ix + _affine_map_[5] * iy + _affine_map_[6] * iz + _affine_map_[7];
            z = _affine_map_[8] * ix + _affine_map_[9] * iy + _affine_map_[10] * iz + _affine_map_[11];
        }
        else {
            if (is_displacement) {
                // get grid coordinate
                ix = w * (grid_xmax - grid_xmin) / (W-1) + grid_xmin;
                iy = h * (grid_ymax - grid_ymin) / (H-1) + grid_ymin;
                iz = d * (grid_zmax - grid_zmin) / (D-1) + grid_zmin;
                pax = ix; pay = iy; paz = iz;
                // apply affine matrix
                if(affine_3d) {
                    x = _affine_map_[0] * ix + _affine_map_[1] * iy + _affine_map_[2] * iz + _affine_map_[3];
                    y = _affine_map_[4] * ix + _affine_map_[5] * iy + _affine_map_[6] * iz + _affine_map_[7];
                    z = _affine_map_[8] * ix + _affine_map_[9] * iy + _affine_map_[10] * iz + _affine_map_[11];
                }
                else {
                    x = ix; y = iy; z = iz;
                }
                // add to displacement
                if (grid) {
                    const index_t grid_size = 3 * D * H * W * (broadcast_grid ? 1 : N);
                    if (grid_offset + 2 < grid_size) {
                        x += grid[grid_offset];
                        y += grid[grid_offset + 1];
                        z += grid[grid_offset + 2];
                    }
                }
            }
            else {
                // just get warp, here we wont need affine (and therefore no need of pax, pay, paz)
                if (grid) {
                    const index_t grid_size = 3 * D * H * W * (broadcast_grid ? 1 : N);
                    if (grid_offset + 2 < grid_size) {
                        x = grid[grid_offset];
                        y = grid[grid_offset + 1];
                        z = grid[grid_offset + 2];
                    }
                }
            }
        }

        // multipliers for gradients on ix, iy, and iz
        gridmath_t gix_mult, giy_mult, giz_mult;
        ix = grid_sampler_compute_source_index_set_grad(x, Wi, padding_mode, align_corners, &gix_mult);
        iy = grid_sampler_compute_source_index_set_grad(y, Hi, padding_mode, align_corners, &giy_mult);
        iz = grid_sampler_compute_source_index_set_grad(z, Di, padding_mode, align_corners, &giz_mult);

        if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
            // get corner pixel values from (x, y, z)
            // for 4d, we used north-east-south-west
            // for 5d, we add top-bottom
            index_t ix_tnw = static_cast<index_t>(std::floor(ix));
            index_t iy_tnw = static_cast<index_t>(std::floor(iy));
            index_t iz_tnw = static_cast<index_t>(std::floor(iz));

            index_t ix_tne = ix_tnw + 1;
            index_t iy_tne = iy_tnw;
            index_t iz_tne = iz_tnw;

            index_t ix_tsw = ix_tnw;
            index_t iy_tsw = iy_tnw + 1;
            index_t iz_tsw = iz_tnw;

            index_t ix_tse = ix_tnw + 1;
            index_t iy_tse = iy_tnw + 1;
            index_t iz_tse = iz_tnw;

            index_t ix_bnw = ix_tnw;
            index_t iy_bnw = iy_tnw;
            index_t iz_bnw = iz_tnw + 1;

            index_t ix_bne = ix_tnw + 1;
            index_t iy_bne = iy_tnw;
            index_t iz_bne = iz_tnw + 1;

            index_t ix_bsw = ix_tnw;
            index_t iy_bsw = iy_tnw + 1;
            index_t iz_bsw = iz_tnw + 1;

            index_t ix_bse = ix_tnw + 1;
            index_t iy_bse = iy_tnw + 1;
            index_t iz_bse = iz_tnw + 1;

            // get surfaces to each neighbor:
            gridmath_t tnw = (ix_bse - ix)        * (iy_bse - iy)        * (iz_bse - iz);
            gridmath_t tne = (ix        - ix_bsw) * (iy_bsw - iy)        * (iz_bsw - iz);
            gridmath_t tsw = (ix_bne - ix)        * (iy        - iy_bne) * (iz_bne - iz);
            gridmath_t tse = (ix        - ix_bnw) * (iy        - iy_bnw) * (iz_bnw - iz);
            gridmath_t bnw = (ix_tse - ix)        * (iy_tse - iy)        * (iz - iz_tse);
            gridmath_t bne = (ix        - ix_tsw) * (iy_tsw - iy)        * (iz - iz_tsw);
            gridmath_t bsw = (ix_tne - ix)        * (iy        - iy_tne) * (iz - iz_tne);
            gridmath_t bse = (ix        - ix_tnw) * (iy        - iy_tnw) * (iz - iz_tnw);

            gridmath_t gix = static_cast<gridmath_t>(0), giy = static_cast<gridmath_t>(0), giz = static_cast<gridmath_t>(0);
            // get grad_output pointer
            const scalar_t *gOut_ptr_NCDHW = grad_output + w + W * (h + H * (d + D * (C * n)));

            // offset for grad_input
            index_t NC_offset = (broadcast_input ? 0 : (n * C * Hi * Wi * Di));
            // if grad_input is provided, we need to offset the input pointer
            const scalar_t *inp_ptr_NC = input + NC_offset;
            // get offsets to add
            const index_t inp_sC = Hi * Wi * Di;
            const index_t gInp_sC = Hi * Wi * Di;
            const index_t gOut_sC = H * W * D;
            const index_t grad_input_memory_span = (broadcast_input ? 1 : N) * (C * Hi * Wi * Di);
            // calculate bilinear weighted pixel value and set output pixel
            for (index_t c = 0; c < C; ++c, gOut_ptr_NCDHW += gOut_sC, NC_offset += gInp_sC, inp_ptr_NC += inp_sC) {
                scalar_t gOut = *gOut_ptr_NCDHW;
                // calculate and set grad_input. See Note [Passing pointer and offset to fastAtomicAdd].
                if (grad_input) {
                    safe_add_3d_oneoffset(grad_input, iz_tnw, iy_tnw, ix_tnw, Di, Hi, Wi, static_cast<scalar_t>(tnw) * gOut, NC_offset, grad_input_memory_span);
                    safe_add_3d_oneoffset(grad_input, iz_tne, iy_tne, ix_tne, Di, Hi, Wi, static_cast<scalar_t>(tne) * gOut, NC_offset, grad_input_memory_span);
                    safe_add_3d_oneoffset(grad_input, iz_tsw, iy_tsw, ix_tsw, Di, Hi, Wi, static_cast<scalar_t>(tsw) * gOut, NC_offset, grad_input_memory_span);
                    safe_add_3d_oneoffset(grad_input, iz_tse, iy_tse, ix_tse, Di, Hi, Wi, static_cast<scalar_t>(tse) * gOut, NC_offset, grad_input_memory_span);
                    safe_add_3d_oneoffset(grad_input, iz_bnw, iy_bnw, ix_bnw, Di, Hi, Wi, static_cast<scalar_t>(bnw) * gOut, NC_offset, grad_input_memory_span);
                    safe_add_3d_oneoffset(grad_input, iz_bne, iy_bne, ix_bne, Di, Hi, Wi, static_cast<scalar_t>(bne) * gOut, NC_offset, grad_input_memory_span);
                    safe_add_3d_oneoffset(grad_input, iz_bsw, iy_bsw, ix_bsw, Di, Hi, Wi, static_cast<scalar_t>(bsw) * gOut, NC_offset, grad_input_memory_span);
                    safe_add_3d_oneoffset(grad_input, iz_bse, iy_bse, ix_bse, Di, Hi, Wi, static_cast<scalar_t>(bse) * gOut, NC_offset, grad_input_memory_span);
                }
                // calculate grad_grid
                gridmath_t gOutGMT = static_cast<gridmath_t>(gOut);
                if (within_bounds_3d(iz_tnw, iy_tnw, ix_tnw, Di, Hi, Wi)) {
                    gridmath_t tnw_val = static_cast<gridmath_t>(inp_ptr_NC[ix_tnw + Wi * (iy_tnw + Hi * iz_tnw)]);
                    gix -= tnw_val * (iy_bse - iy)        * (iz_bse - iz)        * gOutGMT;
                    giy -= tnw_val * (ix_bse - ix)        * (iz_bse - iz)        * gOutGMT;
                    giz -= tnw_val * (ix_bse - ix)        * (iy_bse - iy)        * gOutGMT;
                }
                if (within_bounds_3d(iz_tne, iy_tne, ix_tne, Di, Hi, Wi)) {
                    gridmath_t tne_val = static_cast<gridmath_t>(inp_ptr_NC[ix_tne + Wi * (iy_tne + Hi * iz_tne)]);
                    gix += tne_val * (iy_bsw - iy)        * (iz_bsw - iz)        * gOutGMT;
                    giy -= tne_val * (ix        - ix_bsw) * (iz_bsw - iz)        * gOutGMT;
                    giz -= tne_val * (ix        - ix_bsw) * (iy_bsw - iy)        * gOutGMT;
                }
                if (within_bounds_3d(iz_tsw, iy_tsw, ix_tsw, Di, Hi, Wi)) {
                    gridmath_t tsw_val = static_cast<gridmath_t>(inp_ptr_NC[ix_tsw + Wi * (iy_tsw + Hi * iz_tsw)]);
                    gix -= tsw_val * (iy - iy_bne)        * (iz_bne - iz)        * gOutGMT;
                    giy += tsw_val * (ix_bne - ix)        * (iz_bne - iz)        * gOutGMT;
                    giz -= tsw_val * (ix_bne - ix)        * (iy        - iy_bne) * gOutGMT;
                }
                if (within_bounds_3d(iz_tse, iy_tse, ix_tse, Di, Hi, Wi)) {
                    gridmath_t tse_val = static_cast<gridmath_t>(inp_ptr_NC[ix_tse + Wi * (iy_tse + Hi * iz_tse)]);
                    gix += tse_val * (iy - iy_bnw)        * (iz_bnw - iz)        * gOutGMT;
                    giy += tse_val * (ix        - ix_bnw) * (iz_bnw - iz)        * gOutGMT;
                    giz -= tse_val * (ix        - ix_bnw) * (iy        - iy_bnw) * gOutGMT;
                }
                if (within_bounds_3d(iz_bnw, iy_bnw, ix_bnw, Di, Hi, Wi)) {
                    gridmath_t bnw_val = static_cast<gridmath_t>(inp_ptr_NC[ix_bnw + Wi * (iy_bnw + Hi * iz_bnw)]);
                    gix -= bnw_val * (iy_tse - iy)        * (iz - iz_tse)        * gOutGMT;
                    giy -= bnw_val * (ix_tse - ix)        * (iz - iz_tse)        * gOutGMT;
                    giz += bnw_val * (ix_tse - ix)        * (iy_tse - iy)        * gOutGMT;
                }
                if (within_bounds_3d(iz_bne, iy_bne, ix_bne, Di, Hi, Wi)) {
                    gridmath_t bne_val = static_cast<gridmath_t>(inp_ptr_NC[ix_bne + Wi * (iy_bne + Hi * iz_bne)]);
                    gix += bne_val * (iy_tsw - iy)        * (iz - iz_tsw)        * gOutGMT;
                    giy -= bne_val * (ix        - ix_tsw) * (iz - iz_tsw)        * gOutGMT;
                    giz += bne_val * (ix        - ix_tsw) * (iy_tsw - iy)        * gOutGMT;
                }
                if (within_bounds_3d(iz_bsw, iy_bsw, ix_bsw, Di, Hi, Wi)) {
                    gridmath_t bsw_val = static_cast<gridmath_t>(inp_ptr_NC[ix_bsw + Wi * (iy_bsw + Hi * iz_bsw)]);
                    gix -= bsw_val * (iy - iy_tne)        * (iz - iz_tne)        * gOutGMT;
                    giy += bsw_val * (ix_tne - ix)        * (iz - iz_tne)        * gOutGMT;
                    giz += bsw_val * (ix_tne - ix)        * (iy        - iy_tne) * gOutGMT;
                }
                if (within_bounds_3d(iz_bse, iy_bse, ix_bse, Di, Hi, Wi)) {
                    gridmath_t bse_val = static_cast<gridmath_t>(inp_ptr_NC[ix_bse + Wi * (iy_bse + Hi * iz_bse)]);
                    gix += bse_val * (iy - iy_tnw)        * (iz - iz_tnw)        * gOutGMT;
                    giy += bse_val * (ix        - ix_tnw) * (iz - iz_tnw)        * gOutGMT;
                    giz += bse_val * (ix        - ix_tnw) * (iy        - iy_tnw) * gOutGMT;
                }
            }

            // multiply by grad_output multiplier
            gix = gix_mult * gix;
            giy = giy_mult * giy;
            giz = giz_mult * giz;

            // calculate grad_grid
            if (grad_grid) {
                if (broadcast_grid) {
                    index_t grad_grid_index = 3 * (w + W * (h + H * (d)));
                    gpuAtomicAdd(grad_grid + grad_grid_index, static_cast<grid_t>(gix));
                    gpuAtomicAdd(grad_grid + grad_grid_index + 1, static_cast<grid_t>(giy));
                    gpuAtomicAdd(grad_grid + grad_grid_index + 2, static_cast<grid_t>(giz));
                } 
                else {
                    index_t grad_grid_index = 3 * (w + W * (h + H * (d + D * n)));
                    grad_grid[grad_grid_index] = static_cast<grid_t>(gix);
                    grad_grid[grad_grid_index + 1] = static_cast<grid_t>(giy);
                    grad_grid[grad_grid_index + 2] = static_cast<grid_t>(giz);
                }
            }

            // if affine_3d grad is required
            if (grad_affine_collect) {
                // add it to local registers
                _affine_grad_[0] += gix * pax;
                _affine_grad_[1] += gix * pay;
                _affine_grad_[2] += gix * paz;
                _affine_grad_[3] += gix;
                _affine_grad_[4] += giy * pax;
                _affine_grad_[5] += giy * pay;
                _affine_grad_[6] += giy * paz;
                _affine_grad_[7] += giy;
                _affine_grad_[8] += giz * pax;
                _affine_grad_[9] += giz * pay;
                _affine_grad_[10] += giz * paz;
                _affine_grad_[11] += giz;
            }

        } else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
            if (grad_input) {
                auto ix_nearest = static_cast<index_t>(std::nearbyint(ix));
                auto iy_nearest = static_cast<index_t>(std::nearbyint(iy));
                auto iz_nearest = static_cast<index_t>(std::nearbyint(iz));

                // assign nearest neighbour pixel value to output pixel
                const scalar_t *gOut_ptr_NCDHW = grad_output + w + W * (h + H * (d + D * (C * n)));
                index_t NC_offset = (broadcast_input ? 0 : (n * C * Hi * Wi * Di));
                const index_t gInp_sC = Hi * Wi * Di;
                const index_t gOut_sC = H * W * D;
                const index_t grad_input_memory_span = (broadcast_input ? 1 : N) * (C * Hi * Wi * Di);
                // calculate grad_input for nearest neighbour
                for (index_t c = 0; c < C; ++c, gOut_ptr_NCDHW += gOut_sC, NC_offset += gInp_sC) {
                    // calculate and set grad_input. See Note [Passing pointer and offset to fastAtomicAdd].
                    safe_add_3d_oneoffset(grad_input, iz_nearest, iy_nearest, ix_nearest,
                                             Di, Hi, Wi, *gOut_ptr_NCDHW,
                                             NC_offset, grad_input_memory_span);
                }
            }
        }
    }

    // // if affine_3d grad is required
    if (grad_affine_collect) {
        // add it to local registers
        #pragma unroll
        for (int affid = 0; affid < 12; ++affid) {
            // put it in shared memory to compute the sum over the batch dimension
            _affine_grad_shared_[threadIdx.x] = _affine_grad_[affid];
            __syncthreads();
            
            // reduce over threads
            for (int tid = BLOCKSIZE_3D / 2; tid > 0; tid /= 2) {
                if (threadIdx.x < tid) {
                    _affine_grad_shared_[threadIdx.x] += _affine_grad_shared_[threadIdx.x + tid];
                }
                __syncthreads();
            }
            
            // write to global memory
            if (threadIdx.x == 0) {
                // broadcasted, we need to perform safe atomic add to avoid conflicts with other threads along batch dimension
                if (broadcast_affine_3d) {
                    const index_t offset = blockIdx.x*12 + affid;
                    gpuAtomicAdd(grad_affine_collect + offset, static_cast<grid_t>(_affine_grad_shared_[0]));
                }
                else {
                    const index_t offset = affid + 12 * (blockIdx.x + gridDim.x * n);
                    grad_affine_collect[offset] = static_cast<grid_t>(_affine_grad_shared_[0]);
                }
            }
            __syncthreads();
        }
    }
}

torch::Tensor fused_grid_sampler_3d_forward_impl(
    const torch::Tensor &input, 
    const std::optional<torch::Tensor> affine_3d,
    const std::optional<torch::Tensor> grid,
    const int64_t out_D,
    const int64_t out_H,
    const int64_t out_W,
    const float grid_xmin, 
    const float grid_ymin,
    const float grid_zmin,
    const float grid_xmax,
    const float grid_ymax,
    const float grid_zmax,
    const bool is_displacement,
    int64_t interpolation_mode, int64_t padding_mode, bool align_corners) {
    // See NOTE [ grid_sampler Native Functions ].
    // Add checks here in case this is called instead of grid_sampler.
    //   check_grid_sampler_common(input, grid);
    //   check_grid_sampler_3d(input, grid, interpolation_mode);

    int64_t D, H, W;

    TORCH_CHECK(input.dim() == 5, "input must be 5D");
    TORCH_CHECK(input.device().is_cuda(), "input must be on CUDA");
    TORCH_CHECK(input.is_contiguous(), "input must be contiguous");
    TORCH_CHECK(grid.has_value() || affine_3d.has_value(), "one of grid or affine_3d must exist");

    // device and stream guards
    c10::DeviceGuard guard(input.device());
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream(input.device().index());
    at::cuda::CUDAStreamGuard stream_guard(stream);

    // see if we need to broadcast any variable
    int64_t batch_size_max = input.size(0);
    if (affine_3d.has_value()) {
        batch_size_max = std::max(batch_size_max, affine_3d.value().size(0));
    }
    if (grid.has_value()) {
        batch_size_max = std::max(batch_size_max, grid.value().size(0));
    }
    // broadcast none by default 
    bool broadcast_input = false, broadcast_affine_3d = false, broadcast_grid = false;
    if (batch_size_max > 1) {
        if (input.size(0) == 1) {
            broadcast_input = true;
        } else if (input.size(0) != batch_size_max) {
            TORCH_CHECK(false, "input batch size must match batch size of affine_3d or grid");
        }

        // broadcast affine_3d if it exists
        if (affine_3d.has_value() && affine_3d.value().size(0) == 1) {
            broadcast_affine_3d = true;
        } else if (affine_3d.has_value() && affine_3d.value().size(0) != batch_size_max) {  
            TORCH_CHECK(false, "affine_3d batch size must match batch size of input or grid");
        }
        
        // broadcast grid if it exists
        if (grid.has_value() && grid.value().size(0) == 1) {
            broadcast_grid = true;
        } else if (grid.has_value() && grid.value().size(0) != batch_size_max) {
            TORCH_CHECK(false, "grid batch size must match batch size of input or affine_3d");
        }
    }

    // D, H, W will be determined by grid
    if (grid.has_value()) {
        check_grid_sampler_common_v2(input, grid.value());
        check_grid_sampler_3d(input, grid.value(), interpolation_mode);
        TORCH_CHECK(grid.value().is_contiguous(), "grid must be contiguous");
        D = grid.value().size(1);
        H = grid.value().size(2);
        W = grid.value().size(3);
    }
    else {
        // grid does not exist, affine must exist
        // size of output is determined by input (coordinates determined by affine)
        D = out_D;
        H = out_H;
        W = out_W;
    }

    if (affine_3d.has_value()) {
        // TORCH_CHECK(input.size(0) == affine_3d.value().size(0), "input and affine_3d must have the same batch size");
        TORCH_CHECK(affine_3d.value().dim() == 3, "affine_3d must be (B, 3, 4)");
        TORCH_CHECK(affine_3d.value().device().is_cuda(), "affine_3d must be on CUDA");
        TORCH_CHECK(affine_3d.value().is_contiguous(), "affine_3d must be contiguous");
        TORCH_CHECK(affine_3d.value().size(1) == 3, "affine_3d must be (B, 3, 4)");
        TORCH_CHECK(affine_3d.value().size(2) == 4, "affine_3d must be (B, 3, 4)");
    }

    // define output
    int64_t N = batch_size_max;
    int64_t C = input.size(1);
    torch::Tensor output = torch::zeros({batch_size_max, C, D, H, W}, input.options());

    // input size parameters
    int64_t count = N * D * H * W;

    // input spatial size parameters
    int64_t Di = input.size(2);
    int64_t Hi = input.size(3);
    int64_t Wi = input.size(4);

    // get scalar type of grid/affine
    auto grid_scalar_type = input.scalar_type();
    if (grid.has_value()) {
        grid_scalar_type = grid.value().scalar_type();
    }
    else {
        grid_scalar_type = affine_3d.value().scalar_type();
    }

    if (count > 0) {
        AT_DISPATCH_FLOATING_TYPES_AND(
        at::ScalarType::BFloat16,
        input.scalar_type(), "fused_grid_sampler_3d_forward_kernel", [&] {
            using input_t = scalar_t;
            // check if grid is 32-bit
            AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::BFloat16,
            grid_scalar_type, "fused_grid_sampler_3d_forward_kernel_sub", [&] {
                using grid_t = scalar_t;  // reuse macro variable name here

                bool grid32bit;
                if(grid.has_value()) {
                    grid32bit = canUse32BitIndexMath(grid.value());
                } else {
                    grid32bit = true;
                }
                if (canUse32BitIndexMath(input) && grid32bit &&
                    canUse32BitIndexMath(output)) {
                    fused_grid_sampler_3d_forward_kernel<input_t, grid_t>
                    <<<GET_BLOCKS(count, 512), 512, 0, stream>>>(
                        static_cast<int>(count),
                        input.data_ptr<input_t>(),
                        grid.has_value() ? grid.value().data_ptr<grid_t>() : nullptr,
                        affine_3d.has_value() ? affine_3d.value().data_ptr<grid_t>() : nullptr,
                        static_cast<int>(N), static_cast<int>(C), static_cast<int>(Di), static_cast<int>(Hi), static_cast<int>(Wi),
                        static_cast<int>(D), static_cast<int>(H), static_cast<int>(W),
                        grid_xmin, grid_ymin, grid_zmin, grid_xmax, grid_ymax, grid_zmax,
                        is_displacement,
                        // output
                        output.data_ptr<input_t>(),
                        static_cast<GridSamplerInterpolation>(interpolation_mode),
                        static_cast<GridSamplerPadding>(padding_mode),
                        align_corners,
                        broadcast_input,
                        broadcast_affine_3d,
                        broadcast_grid
                    );
                    C10_CUDA_KERNEL_LAUNCH_CHECK();
                } else {
                    fused_grid_sampler_3d_forward_kernel<input_t, grid_t>
                    <<<GET_BLOCKS(count, 512), 512, 0, stream>>>(
                        count,
                        input.data_ptr<input_t>(),
                        grid.has_value() ? grid.value().data_ptr<grid_t>() : nullptr,
                        affine_3d.has_value() ? affine_3d.value().data_ptr<grid_t>() : nullptr,
                        N, C, Di, Hi, Wi,
                        D, H, W,
                        grid_xmin, grid_ymin, grid_zmin, grid_xmax, grid_ymax, grid_zmax,
                        is_displacement,
                        output.data_ptr<input_t>(),
                        static_cast<GridSamplerInterpolation>(interpolation_mode),
                        static_cast<GridSamplerPadding>(padding_mode),
                        align_corners,
                        broadcast_input,
                        broadcast_affine_3d,
                        broadcast_grid
                        );
                    C10_CUDA_KERNEL_LAUNCH_CHECK();
                }
            });
        });
    }
    return output;
}

void fused_grid_sampler_3d_backward_impl(
        /* we need input, A, u */
        const torch::Tensor &input, 
        const std::optional<torch::Tensor> affine_3d,
        const std::optional<torch::Tensor> grid,
        /* we need grad_output, grad_input, grad_affine, grad_grid, some may be empty or zeros */
        const torch::Tensor &grad_output,
        const std::optional<torch::Tensor> &grad_input,
        const std::optional<torch::Tensor> &grad_affine,
        const std::optional<torch::Tensor> &grad_grid,
        /* input parameters = output size, grid bounds, is_displacement, interpolation_mode, padding_mode, align_corners */
        const int64_t out_D,
        const int64_t out_H,
        const int64_t out_W,
        const float grid_xmin, 
        const float grid_ymin,
        const float grid_zmin,
        const float grid_xmax,
        const float grid_ymax,
        const float grid_zmax,
        const bool is_displacement,
        int64_t interpolation_mode, int64_t padding_mode, bool align_corners) {

        int64_t D, H, W;
        TORCH_CHECK(input.dim() == 5, "input must be 5D");
        TORCH_CHECK(input.device().is_cuda(), "input must be on CUDA");
        TORCH_CHECK(input.is_contiguous(), "input must be contiguous");
        TORCH_CHECK(grid.has_value() || affine_3d.has_value(), "one of grid or affine_3d must exist");
        TORCH_CHECK(grad_input.has_value() || grad_affine.has_value() || grad_grid.has_value(), "at least one of grad_input, grad_affine, grad_grid must exist");

        // device and stream guards
        c10::DeviceGuard guard(input.device());
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream(input.device().index());
        at::cuda::CUDAStreamGuard stream_guard(stream);

        // see if we need to broadcast any variable
        int64_t batch_size_max = input.size(0);
        if (affine_3d.has_value()) {
            batch_size_max = std::max(batch_size_max, affine_3d.value().size(0));
        }
        if (grid.has_value()) {
            batch_size_max = std::max(batch_size_max, grid.value().size(0));
        }

        // broadcast none by default 
        bool broadcast_input = false, broadcast_affine_3d = false, broadcast_grid = false;
        if (batch_size_max > 1) {
            if (input.size(0) == 1) {
                broadcast_input = true;
            } else if (input.size(0) != batch_size_max) {
                TORCH_CHECK(false, "input batch size must match batch size of affine_3d or grid");
            }

            // broadcast affine_3d if it exists
            if (affine_3d.has_value() && affine_3d.value().size(0) == 1) {
                broadcast_affine_3d = true;
            } else if (affine_3d.has_value() && affine_3d.value().size(0) != batch_size_max) {  
                TORCH_CHECK(false, "affine_3d batch size must match batch size of input or grid");
            }
            
            // broadcast grid if it exists
            if (grid.has_value() && grid.value().size(0) == 1) {
                broadcast_grid = true;
            } else if (grid.has_value() && grid.value().size(0) != batch_size_max) {
                TORCH_CHECK(false, "grid batch size must match batch size of input or affine_3d");
            }
        }

        // determine if we need to compute gradients
        bool input_requires_grad = grad_input.has_value();
        bool affine_requires_grad = grad_affine.has_value() && affine_3d.has_value();
        bool grid_requires_grad = grad_grid.has_value() && grid.has_value();
        // if interpolation mode is nearest, we do not need to compute gradients of grid and affine
        if (static_cast<GridSamplerInterpolation>(interpolation_mode) == GridSamplerInterpolation::Nearest) {
            grid_requires_grad = false;
            affine_requires_grad = false;
        }
        // if grid is provided and it is a displacement, there are no gradients w.r.t. affine
        if (grid.has_value() && !is_displacement) {
            affine_requires_grad = false;
        }

        if (!grid_requires_grad && !affine_requires_grad && !input_requires_grad) {
            // nothing to compute
            return;
        }

        // D, H, W will be determined by grid
        if (grid.has_value()) {
            check_grid_sampler_common_v2(input, grid.value());
            check_grid_sampler_3d(input, grid.value(), interpolation_mode);
            TORCH_CHECK(grid.value().is_contiguous(), "grid must be contiguous");
            D = grid.value().size(1);
            H = grid.value().size(2);
            W = grid.value().size(3);
        }
        else {
            // grid does not exist, affine must exist
            // size of output is determined by input (coordinates determined by affine)
            D = out_D;
            H = out_H;
            W = out_W;
        }

        if (affine_3d.has_value()) {
            // TORCH_CHECK(input.size(0) == affine_3d.value().size(0), "input and affine_3d must have the same batch size");
            TORCH_CHECK(affine_3d.value().dim() == 3, "affine_3d must be (B, 3, 4)");
            TORCH_CHECK(affine_3d.value().device().is_cuda(), "affine_3d must be on CUDA");
            TORCH_CHECK(affine_3d.value().is_contiguous(), "affine_3d must be contiguous");
            TORCH_CHECK(affine_3d.value().size(1) == 3, "affine_3d must be (B, 3, 4)");
            TORCH_CHECK(affine_3d.value().size(2) == 4, "affine_3d must be (B, 3, 4)");
        }

        // define output
        int64_t N = batch_size_max;
        int64_t C = input.size(1);

        // input size parameters (put batch in a separate dimension)
        int64_t count = D * H * W;

        // input spatial size parameters
        int64_t Di = input.size(2);
        int64_t Hi = input.size(3);
        int64_t Wi = input.size(4);

        // initialize grid and dim
        dim3 blockSize3(BLOCKSIZE_3D, 1, 1);
        int64_t gridSize = GET_BLOCKS(count, BLOCKSIZE_3D);
        gridSize = std::min(gridSize, static_cast<int64_t>(65536));
        dim3 gridSize3(gridSize, batch_size_max, 1);

        // intermediate grad affine collector
        torch::Tensor grad_affine_collect;
        if (affine_requires_grad) {
            grad_affine_collect = torch::zeros({affine_3d.value().size(0), gridSize, 3, 4}, grad_affine.value().options());
        }

        auto grid_scalar_type = grad_output.scalar_type();
        if (grid.has_value()) {
            grid_scalar_type = grid.value().scalar_type();
        }
        else {
            grid_scalar_type = affine_3d.value().scalar_type();
        }

        if (count > 0) {
            AT_DISPATCH_FLOATING_TYPES_AND(
            at::ScalarType::BFloat16,
            input.scalar_type(), "fused_grid_sampler_3d_backward_kernel", [&] {
                // check if grid is 32-bit
                using input_t = scalar_t;
                AT_DISPATCH_FLOATING_TYPES_AND(
                at::ScalarType::BFloat16, grid_scalar_type, "fused_grid_sampler_3d_backward_kernel_sub", [&] {
                    using grid_t = scalar_t;
                    bool grid32bit;
                    if(grid.has_value()) {
                        grid32bit = canUse32BitIndexMath(grid.value());
                    } else {
                        grid32bit = true;
                    }
                    if (canUse32BitIndexMath(input) && grid32bit &&
                        canUse32BitIndexMath(grad_output)) {
                        fused_grid_sampler_3d_backward_kernel<input_t, grid_t>
                        <<<gridSize3, blockSize3, 0, stream>>>(
                            static_cast<int>(count),
                            input.data_ptr<input_t>(),
                            grid.has_value() ? grid.value().data_ptr<grid_t>() : nullptr,
                            affine_3d.has_value() ? affine_3d.value().data_ptr<grid_t>() : nullptr,
                            // grads
                            grad_output.data_ptr<input_t>(),
                            input_requires_grad ? grad_input.value().data_ptr<input_t>() : nullptr,
                            affine_requires_grad ? grad_affine_collect.data_ptr<grid_t>() : nullptr,
                            grid_requires_grad ? grad_grid.value().data_ptr<grid_t>() : nullptr,
                            // input size parameters
                            static_cast<int>(N), static_cast<int>(C), static_cast<int>(Di), static_cast<int>(Hi), static_cast<int>(Wi),
                            static_cast<int>(D), static_cast<int>(H), static_cast<int>(W),
                            grid_xmin, grid_ymin, grid_zmin, grid_xmax, grid_ymax, grid_zmax,
                            is_displacement,
                            // additional parameters
                            static_cast<GridSamplerInterpolation>(interpolation_mode),
                            static_cast<GridSamplerPadding>(padding_mode),
                            align_corners,
                            broadcast_input,
                            broadcast_affine_3d,
                            broadcast_grid
                        );
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                    } else {
                        fused_grid_sampler_3d_backward_kernel<input_t, grid_t>
                        <<<gridSize3, blockSize3, 0, stream>>>(
                            count,
                            input.data_ptr<input_t>(),
                            grid.has_value() ? grid.value().data_ptr<grid_t>() : nullptr,
                            affine_3d.has_value() ? affine_3d.value().data_ptr<grid_t>() : nullptr,
                            // grads
                            grad_output.data_ptr<input_t>(),
                            input_requires_grad ? grad_input.value().data_ptr<input_t>() : nullptr,
                            affine_requires_grad ? grad_affine_collect.data_ptr<grid_t>() : nullptr,
                            grid_requires_grad ? grad_grid.value().data_ptr<grid_t>() : nullptr,
                            // input size parameters
                            N, C, Di, Hi, Wi,
                            D, H, W,
                            grid_xmin, grid_ymin, grid_zmin, grid_xmax, grid_ymax, grid_zmax,
                            is_displacement,
                            // additional parameters
                            static_cast<GridSamplerInterpolation>(interpolation_mode),
                            static_cast<GridSamplerPadding>(padding_mode),
                            align_corners,
                            broadcast_input,
                            broadcast_affine_3d,
                            broadcast_grid
                        );
                        C10_CUDA_KERNEL_LAUNCH_CHECK();
                    }
                });
        });
    }

    if (affine_requires_grad) {
        // sum over the batch dimension
        grad_affine.value().copy_(grad_affine_collect.sum(1));
    }
}
